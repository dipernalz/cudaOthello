#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <unistd.h>

#include <iostream>

#include "constants.hpp"
#include "mcts.hpp"
#include "othello.hpp"

hiprandGenerator_t rng;

int main(int ac, char **av) {
    srand((unsigned)time(NULL) ^ getpid());

    othello *board = new othello(av[1], *av[2]);

    switch (*av[3]) {
        // random move
        case '0': {
            vector<move> *moves = board->generate_moves();
            if (!moves->is_empty()) {
                board->make_move(moves->get_random_entry());
                board->print();
            }
            delete moves;
            break;
        }
        // MCTS move
        case '1': {
            find_best_move(board, false);
            break;
        }
        // make given move
        case '2': {
            board->make_move({(int8_t)atoi(av[4]), (int8_t)atoi(av[5])});
            board->print();
        }
        // MCTS with CUDA move
        case '3': {
            hipSetDevice(2);
            hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_MTGP32);
            hiprandSetPseudoRandomGeneratorSeed(rng,
                                               (unsigned)time(NULL) ^ getpid());
            find_best_move(board, true);
            hiprandDestroyGenerator(rng);
            break;
        }
    }

    delete board;
}
