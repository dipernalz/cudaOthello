#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "constants.hpp"
#include "mcts.hpp"
#include "othello.hpp"

move choose_move(othello *board, vector<move> *&moves) {
    for (uint8_t i = 0; i < moves->size(); i++) {
        move mv = moves->get(i);
        if ((mv.row == 0 || mv.row == N - 1) &&
            (mv.col == 0 || mv.col == N - 1)) {
            return mv;
        }
    }

    move mv = moves->get(rand() % moves->size());
    for (uint8_t i = 0; i < 4; i++) {
        if ((mv.row == 1 || mv.row == N - 2) &&
            (mv.col == 1 || mv.col == N - 2))
            mv = moves->get(rand() % moves->size());
        else
            break;
    }
    return mv;
}

static __device__ int is_move(int *black, int *white, int turn, int row,
                              int col) {
    int *board_a, *board_b;
    if (turn == BLACK) {
        board_a = black;
        board_b = white;
    } else {
        board_a = white;
        board_b = black;
    }

    // #pragma unroll
    for (int dr = -1; dr <= 1; dr++) {
        // #pragma unroll
        for (int dc = -1; dc <= 1; dc++) {
            if (dr == 0 && dc == 0) continue;
            int tr = row + dr;
            int tc = col + dc;
            while (POS_IN_BOARD(tr, tc) && board_b[BOARD_INDEX(tr, tc)]) {
                tr += dr;
                tc += dc;
            }
            if (POS_IN_BOARD(tr, tc) && (tr != row + dr || tc != col + dc) &&
                board_a[BOARD_INDEX(tr, tc)])
                return true;
        }
    }

    return false;
}

static __device__ int generate_moves(int *black, int *white, int turn,
                                     int *moves_r, int *moves_c) {
    int n_moves = 0;
    // #pragma unroll
    for (int r = 0; r < N; r++) {
        // #pragma unroll
        for (int c = 0; c < N; c++) {
            if (!black[BOARD_INDEX(r, c)] && !white[BOARD_INDEX(r, c)] &&
                is_move(black, white, turn, r, c)) {
                moves_r[n_moves] = r;
                moves_c[n_moves] = c;
                n_moves++;
            }
        }
    }
    return n_moves;
}

static __device__ void make_move(int *black, int *white, int *turn,
                                 int *n_black, int *n_white, int row, int col) {
    int *board_a, *board_b;
    int *n_color_a, *n_color_b;
    if (*turn == BLACK) {
        board_a = black;
        board_b = white;
        n_color_a = n_black;
        n_color_b = n_white;
    } else {
        board_a = white;
        board_b = black;
        n_color_a = n_white;
        n_color_b = n_black;
    }

    int move_made = false;
    // #pragma unroll
    for (int dr = -1; dr <= 1; dr++) {
        // #pragma unroll
        for (int dc = -1; dc <= 1; dc++) {
            if (dr == 0 && dc == 0) continue;
            int tr = row + dr;
            int tc = col + dc;
            while (POS_IN_BOARD(tr, tc) && board_b[BOARD_INDEX(tr, tc)]) {
                tr += dr;
                tc += dc;
            }
            if (POS_IN_BOARD(tr, tc) && (tr != row + dr || tc != col + dc) &&
                board_a[BOARD_INDEX(tr, tc)]) {
                *n_color_a -= move_made;
                move_made = true;
                do {
                    tr -= dr;
                    tc -= dc;
                    board_a[BOARD_INDEX(tr, tc)] = TAKEN;
                    board_b[BOARD_INDEX(tr, tc)] = EMPTY;
                    (*n_color_a)++;
                    (*n_color_b)--;
                } while (tr != row || tc != col);
                (*n_color_b)++;
            }
        }
    }

    *turn = !*turn;
}

static __global__ void sim_rand_game(int *black, int *white, int *turn,
                                     int *n_black, int *n_white, int *result) {
    __shared__ int s_black[64];
    __shared__ int s_white[64];
    __shared__ int s_moves_r[64];
    __shared__ int s_moves_c[64];

    int r_turn = *turn;
    int r_n_black = *n_black;
    int r_n_white = *n_white;

#pragma unroll
    for (int i = 0; i < N * N; i++) {
        s_black[i] = black[i];
        s_white[i] = white[i];
    }

    int *piece_count_a, *piece_count_b;
    if (r_turn == BLACK) {
        piece_count_a = &r_n_black;
        piece_count_b = &r_n_white;
    } else {
        piece_count_a = &r_n_white;
        piece_count_b = &r_n_black;
    }

    while (r_n_black + r_n_white != N * N) {
        int n_moves =
            generate_moves(s_black, s_white, r_turn, s_moves_r, s_moves_c);
        if (n_moves == 0) {
            r_turn = !r_turn;
            n_moves =
                generate_moves(s_black, s_white, r_turn, s_moves_r, s_moves_c);
            if (n_moves == 0) break;
            make_move(s_black, s_white, &r_turn, &r_n_black, &r_n_white,
                      s_moves_r[0], s_moves_c[0]);
            continue;
        }
        make_move(s_black, s_white, &r_turn, &r_n_black, &r_n_white,
                  s_moves_r[0], s_moves_c[0]);
    }

    *result = *piece_count_a - *piece_count_b;
}

void reset_cuda_board(int *h_black, int *h_white, int *h_turn, int *h_n_black,
                      int *h_n_white, int *c_black, int *c_white, int *c_turn,
                      int *c_n_black, int *c_n_white) {
    hipMemcpy(c_black, h_black, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_white, h_white, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_turn, h_turn, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_n_black, h_n_black, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_n_white, h_n_white, sizeof(int), hipMemcpyHostToDevice);
}

sim_results sim_n_games_cuda(uint32_t n, othello *board) {
    sim_results results = {n, 0, 0};

    int *h_black, *h_white, h_turn, h_n_black, h_n_white, h_result;
    h_black = (int *)malloc(N * N * sizeof(int));
    h_white = (int *)malloc(N * N * sizeof(int));
    for (int i = 0; i < N * N; i++) {
        h_black[i] = (int)board->black[i];
        h_white[i] = (int)board->white[i];
        h_turn = (int)board->turn;
        h_n_black = (int)board->n_black;
        h_n_white = (int)board->n_white;
    }

    int *c_black, *c_white, *c_turn, *c_n_black, *c_n_white, *c_result;
    hipMalloc(&c_black, N * N * sizeof(int));
    hipMalloc(&c_white, N * N * sizeof(int));
    hipMalloc(&c_turn, sizeof(int));
    hipMalloc(&c_n_black, sizeof(int));
    hipMalloc(&c_n_white, sizeof(int));
    hipMalloc(&c_result, sizeof(int));

    for (uint32_t i = 0; i < n; i++) {
        reset_cuda_board(h_black, h_white, &h_turn, &h_n_black, &h_n_white,
                         c_black, c_white, c_turn, c_n_black, c_n_white);
        sim_rand_game<<<1, 1>>>(c_black, c_white, c_turn, c_n_black, c_n_white,
                                c_result);
        hipMemcpy(&h_result, c_result, sizeof(int), hipMemcpyDeviceToHost);
        if (h_result > 0)
            results.wins++;
        else if (h_result < 0)
            results.losses++;
        break;
    }

    hipFree(c_black);
    hipFree(c_white);
    hipFree(c_turn);
    hipFree(c_n_black);
    hipFree(c_n_white);
    hipFree(c_result);

    return results;
}
