#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "constants.hpp"
#include "mcts.hpp"
#include "othello.hpp"

#define B 8
#define T 8

extern hiprandGenerator_t rng;

static __device__ int is_move(int *black, int *white, int turn, int row,
                              int col) {
    int *board_a, *board_b;
    if (turn == BLACK) {
        board_a = black;
        board_b = white;
    } else {
        board_a = white;
        board_b = black;
    }

    for (int dr = -1; dr <= 1; dr++) {
        for (int dc = -1; dc <= 1; dc++) {
            if (dr == 0 && dc == 0) continue;
            int tr = row + dr;
            int tc = col + dc;
            while (POS_IN_BOARD(tr, tc) && board_b[BOARD_INDEX(tr, tc)]) {
                tr += dr;
                tc += dc;
            }
            if (POS_IN_BOARD(tr, tc) && (tr != row + dr || tc != col + dc) &&
                board_a[BOARD_INDEX(tr, tc)])
                return true;
        }
    }

    return false;
}

static __device__ int generate_moves(int *black, int *white, int turn,
                                     int *moves_r, int *moves_c) {
    int n_moves = 0;
    for (int r = 0; r < N; r++) {
        for (int c = 0; c < N; c++) {
            if (!black[BOARD_INDEX(r, c)] && !white[BOARD_INDEX(r, c)] &&
                is_move(black, white, turn, r, c)) {
                moves_r[n_moves] = r;
                moves_c[n_moves] = c;
                n_moves++;
            }
        }
    }
    return n_moves;
}

static __device__ void make_move(int *black, int *white, int *turn,
                                 int *n_black, int *n_white, int row, int col) {
    int *board_a, *board_b;
    int *n_color_a, *n_color_b;
    if (*turn == BLACK) {
        board_a = black;
        board_b = white;
        n_color_a = n_black;
        n_color_b = n_white;
    } else {
        board_a = white;
        board_b = black;
        n_color_a = n_white;
        n_color_b = n_black;
    }

    int move_made = false;
    for (int dr = -1; dr <= 1; dr++) {
        for (int dc = -1; dc <= 1; dc++) {
            if (dr == 0 && dc == 0) continue;
            int tr = row + dr;
            int tc = col + dc;
            while (POS_IN_BOARD(tr, tc) && board_b[BOARD_INDEX(tr, tc)]) {
                tr += dr;
                tc += dc;
            }
            if (POS_IN_BOARD(tr, tc) && (tr != row + dr || tc != col + dc) &&
                board_a[BOARD_INDEX(tr, tc)]) {
                *n_color_a -= move_made;
                move_made = true;
                do {
                    tr -= dr;
                    tc -= dc;
                    board_a[BOARD_INDEX(tr, tc)] = TAKEN;
                    board_b[BOARD_INDEX(tr, tc)] = EMPTY;
                    (*n_color_a)++;
                    (*n_color_b)--;
                } while (tr != row || tc != col);
                (*n_color_b)++;
            }
        }
    }

    *turn = !*turn;
}

static __global__ void sim_rand_game(int *black, int *white, unsigned int *rand,
                                     int *turn, int *n_black, int *n_white,
                                     int *wins, int *losses) {
    __shared__ int s_black[T][N * N];
    __shared__ int s_white[T][N * N];
    __shared__ int s_moves_r[T][N * N];
    __shared__ int s_moves_c[T][N * N];

    int r_turn = *turn;
    int r_n_black = *n_black;
    int r_n_white = *n_white;
    int r_rand_idx = T * N * N * blockIdx.x + N * N * threadIdx.x;

    for (int i = 0; i < N * N; i++) {
        s_black[threadIdx.x][i] = black[i];
        s_white[threadIdx.x][i] = white[i];
    }

    int *piece_count_a, *piece_count_b;
    if (r_turn == BLACK) {
        piece_count_a = &r_n_black;
        piece_count_b = &r_n_white;
    } else {
        piece_count_a = &r_n_white;
        piece_count_b = &r_n_black;
    }

    int idx;
    while (r_n_black + r_n_white != N * N) {
        int n_moves =
            generate_moves(s_black[threadIdx.x], s_white[threadIdx.x], r_turn,
                           s_moves_r[threadIdx.x], s_moves_c[threadIdx.x]);
        if (n_moves == 0) {
            r_turn = !r_turn;
            n_moves = generate_moves(s_black[threadIdx.x], s_white[threadIdx.x],
                                     r_turn, s_moves_r[threadIdx.x],
                                     s_moves_c[threadIdx.x]);
            if (n_moves == 0) break;
            idx = rand[r_rand_idx++] % n_moves;
            make_move(s_black[threadIdx.x], s_white[threadIdx.x], &r_turn,
                      &r_n_black, &r_n_white, s_moves_r[threadIdx.x][idx],
                      s_moves_c[threadIdx.x][idx]);
            continue;
        }
        idx = rand[r_rand_idx++] % n_moves;
        make_move(s_black[threadIdx.x], s_white[threadIdx.x], &r_turn,
                  &r_n_black, &r_n_white, s_moves_r[threadIdx.x][idx],
                  s_moves_c[threadIdx.x][idx]);
    }

    atomicAdd(wins, *piece_count_a - *piece_count_b > 0 ? 1 : 0);
    atomicAdd(losses, *piece_count_a - *piece_count_b < 0 ? 1 : 0);
}

sim_results sim_games_cuda(othello *board) {
    sim_results results = {B * T, 0, 0};

    int *h_black, *h_white, h_turn, h_n_black, h_n_white;
    h_black = (int *)malloc(N * N * sizeof(int));
    h_white = (int *)malloc(N * N * sizeof(int));
    for (int i = 0; i < N * N; i++) {
        h_black[i] = (int)board->black[i];
        h_white[i] = (int)board->white[i];
        h_turn = (int)board->turn;
        h_n_black = (int)board->n_black;
        h_n_white = (int)board->n_white;
    }

    int *c_black, *c_white, *c_turn, *c_n_black, *c_n_white, *c_wins, *c_losses;
    unsigned int *c_rand;

    hipMalloc(&c_black, N * N * sizeof(int));
    hipMalloc(&c_white, N * N * sizeof(int));
    hipMalloc(&c_rand, B * T * N * N * sizeof(unsigned int));
    hipMalloc(&c_turn, sizeof(int));
    hipMalloc(&c_n_black, sizeof(int));
    hipMalloc(&c_n_white, sizeof(int));
    hipMalloc(&c_wins, sizeof(int));
    hipMalloc(&c_losses, sizeof(int));

    hipMemcpy(c_black, h_black, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_white, h_white, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_turn, &h_turn, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_n_black, &h_n_black, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_n_white, &h_n_white, sizeof(int), hipMemcpyHostToDevice);
    hipMemset(c_wins, 0, sizeof(int));
    hipMemset(c_losses, 0, sizeof(int));

    hiprandGenerate(rng, c_rand, B * T * N * N);
    sim_rand_game<<<B, T>>>(c_black, c_white, c_rand, c_turn, c_n_black,
                            c_n_white, c_wins, c_losses);
    hipMemcpy(&results.wins, c_wins, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&results.losses, c_losses, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(c_black);
    hipFree(c_white);
    hipFree(c_rand);
    hipFree(c_turn);
    hipFree(c_n_black);
    hipFree(c_n_white);
    hipFree(c_wins);
    hipFree(c_losses);

    free(h_black);
    free(h_white);

    return results;
}
