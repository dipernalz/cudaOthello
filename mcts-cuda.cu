#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "constants.hpp"
#include "mcts.hpp"
#include "othello.hpp"

move choose_move(othello *board, vector<move> *&moves) {
    for (uint8_t i = 0; i < moves->size(); i++) {
        move mv = moves->get(i);
        if ((mv.row == 0 || mv.row == N - 1) &&
            (mv.col == 0 || mv.col == N - 1)) {
            return mv;
        }
    }

    move mv = moves->get(rand() % moves->size());
    for (uint8_t i = 0; i < 4; i++) {
        if ((mv.row == 1 || mv.row == N - 2) &&
            (mv.col == 1 || mv.col == N - 2))
            mv = moves->get(rand() % moves->size());
        else
            break;
    }
    return mv;
}

__global__ void sim_rand_game(int *black, int *white, int *turn, int *n_black,
                              int *n_white, int *result) {
    __shared__ int s_black[64];
    __shared__ int s_white[64];
    int r_turn = *turn;
    int r_n_black = *n_black;
    int r_n_white = *n_white;

#pragma unroll
    for (int i = 0; i < N * N; i++) {
        s_black[i] = black[i];
        s_white[i] = white[i];
    }

    printf("cuda\n");
    for (int r = 0; r < N; r++) {
        for (int c = 0; c < N; c++) {
            printf("%c ", s_black[BOARD_INDEX(r, c)]
                              ? 'B'
                              : (s_white[BOARD_INDEX(r, c)] ? 'W' : '.'));
        }
        printf("\n");
    }

    // othello *board = new othello(starting_board);

    // uint8_t *piece_count_a, *piece_count_b;
    // if (board->get_turn() == BLACK) {
    //     piece_count_a = &board->get_n_black();
    //     piece_count_b = &board->get_n_white();
    // } else {
    //     piece_count_a = &board->get_n_white();
    //     piece_count_b = &board->get_n_black();
    // }

    // while (board->get_n_placed() != N * N) {
    //     vector<move> *moves = board->generate_moves();
    //     if (moves->is_empty()) {
    //         board->change_turn();
    //         delete moves;
    //         vector<move> *moves = board->generate_moves();
    //         if (moves->is_empty()) {
    //             delete moves;
    //             break;
    //         }
    //         board->make_move(choose_move(board, moves));
    //         delete moves;
    //         continue;
    //     }
    //     board->make_move(choose_move(board, moves));
    //     delete moves;
    // }

    // int8_t result = ((int8_t)*piece_count_a) - ((int8_t)*piece_count_b);
    // delete board;
    // return result;
}

void reset_cuda_board(int *h_black, int *h_white, int *h_turn, int *h_n_black,
                      int *h_n_white, int *c_black, int *c_white, int *c_turn,
                      int *c_n_black, int *c_n_white) {
    hipMemcpy(c_black, h_black, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_white, h_white, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_turn, h_turn, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_n_black, h_n_black, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_n_black, h_n_black, sizeof(int), hipMemcpyHostToDevice);
}

sim_results sim_n_games_cuda(uint32_t n, othello *board) {
    sim_results results = {n, 0, 0};

    int *h_black, *h_white, h_turn, h_n_black, h_n_white;
    h_black = (int *)malloc(N * N * sizeof(int));
    h_white = (int *)malloc(N * N * sizeof(int));
    for (int i = 0; i < N * N; i++) {
        h_black[i] = (int)board->black[i];
        h_white[i] = (int)board->white[i];
        h_turn = (int)board->turn;
        h_n_black = (int)board->n_black;
        h_n_white = (int)board->n_white;
    }

    int *c_black, *c_white, *c_turn, *c_n_black, *c_n_white, *c_result;
    hipMalloc(&c_black, N * N * sizeof(int));
    hipMalloc(&c_white, N * N * sizeof(int));
    hipMalloc(&c_turn, sizeof(int));
    hipMalloc(&c_n_black, sizeof(int));
    hipMalloc(&c_n_white, sizeof(int));
    hipMalloc(&c_result, sizeof(int));

    for (uint32_t i = 0; i < n; i++) {
        reset_cuda_board(h_black, h_white, &h_turn, &h_n_black, &h_n_white,
                         c_black, c_white, c_turn, c_n_black, c_n_white);
        sim_rand_game<<<1, 1>>>(c_black, c_white, c_turn, c_n_black, c_n_white,
                                c_result);
        // int8_t result = sim_rand_game(board);
        // if (result > 0)
        //     results.wins++;
        // else if (result < 0)
        //     results.losses++;
    }

    hipFree(c_black);
    hipFree(c_white);
    hipFree(c_turn);
    hipFree(c_n_black);
    hipFree(c_n_white);

    return results;
}
